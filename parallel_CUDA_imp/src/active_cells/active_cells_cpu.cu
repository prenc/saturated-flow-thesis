#include "hip/hip_runtime.h"
#include <iostream>
#include <numeric>
#include <stdlib.h>
#include <stdio.h>
#include <math.h>

#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/unique.h>
#include <thrust/execution_policy.h>

//MODEL PARAMS

#define ROWS 100
#define COLS 100

#define CELL_SIZE_X 10
#define CELL_SIZE_Y 10
#define AREA CELL_SIZE_X*CELL_SIZE_Y

#define THICKNESS 50

#define Syinitial 0.1
#define Kinitial  0.0000125

#define headFixed 50
#define headCalculated 50

#define SIMULATION_ITERATIONS 1000
#define BLOCK_SIZE 16

#define DELTA_T 4000;
double qw = 0.001;

int posSy = ROWS / 2;
int posSx = COLS / 2;

struct CA {
    double *head;
    double *Sy;
    double *K;
    double *Source;
} d_read, d_write;

void allocate_memory();
void init_read_ca();
void perform_simulation_on_GPU();
void write_heads_to_file();
void init_write_head();
void free_allocated_memory();

thrust::device_vector<int> d_active_cells_vector;


static void CheckCudaErrorAux(const char *, unsigned, const char *, hipError_t);

#define CUDA_CHECK_RETURN(value) CheckCudaErrorAux(__FILE__,__LINE__, #value, value)

__global__ void simulation_step_kernel(struct CA d_ca, double *d_write_head, int *ac_array, int ac_array_size) {
    unsigned ac_idx_x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned ac_idx_y = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned ac_idx_g = ac_idx_y * COLS + ac_idx_x;

    if(ac_idx_g < ac_array_size){
        unsigned idx_g = ac_array[ac_idx_g];
        unsigned idx_x = idx_g % COLS;
        unsigned idx_y = idx_g / COLS;
		if (idx_y != 0 && idx_y != ROWS - 1) {
				double Q = 0;
				double diff_head;
				double tmp_t;

				if (idx_x >= 1) {
					diff_head = d_ca.head[idx_g - 1] - d_ca.head[idx_g];
					tmp_t = d_ca.K[idx_g] * THICKNESS;
					Q += diff_head * tmp_t;
				}
				if (idx_y >= 1) {
					diff_head = d_ca.head[(idx_y - 1) * COLS + idx_x] - d_ca.head[idx_g];
					tmp_t = d_ca.K[idx_g] * THICKNESS;
					Q += diff_head * tmp_t;
				}
				if (idx_x + 1 < COLS) {
					diff_head = d_ca.head[idx_g + 1] - d_ca.head[idx_g];
					tmp_t = d_ca.K[idx_g] * THICKNESS;
					Q += diff_head * tmp_t;
				}
				if (idx_y + 1 < ROWS) {
					diff_head = d_ca.head[(idx_y + 1) * COLS + idx_x] - d_ca.head[idx_g];
					tmp_t = d_ca.K[idx_g] * THICKNESS;
					Q += diff_head * tmp_t;
				}

				Q -= d_ca.Source[idx_g];

				double ht1 = Q * DELTA_T;
				double ht2 = AREA * d_ca.Sy[idx_g];

			  d_write_head[idx_g] = d_ca.head[idx_g] + ht1 / ht2;
		}
    }
}
void find_active_cells(){
	thrust::host_vector<int> h_active_cells_vector;
	for(int i = 0; i < ROWS; i++){
		for(int j = 0; j < ROWS; j++){
			int idx_g = j* COLS + i;
			if(d_read.head[idx_g] < headFixed){
				h_active_cells_vector.push_back(idx_g);
				if(i + 1 < ROWS) h_active_cells_vector.push_back(idx_g + 1);
				if(i - 1 >= 0 )   h_active_cells_vector.push_back(idx_g - 1);
				if(j + 1 < COLS) h_active_cells_vector.push_back(idx_g + COLS);
				if(j - 1 >= 0)    h_active_cells_vector.push_back(idx_g - COLS);
			}else if(d_read.Source[idx_g] != 0){
				h_active_cells_vector.push_back(idx_g);
			}
		}
	}
	thrust::host_vector<int> h_active_cells_vector_result;
	thrust::sort(h_active_cells_vector.begin(),h_active_cells_vector.end());
	int previous_value = h_active_cells_vector[0];
	h_active_cells_vector_result.push_back(previous_value);
	for(int i = 1; i< h_active_cells_vector.size(); i++){
		if(h_active_cells_vector[i] != previous_value){
			previous_value = h_active_cells_vector[i];
			h_active_cells_vector_result.push_back(previous_value);
		}
	}
	d_active_cells_vector = h_active_cells_vector_result;
}
int main(void) {
    allocate_memory();

    init_read_ca();

    init_write_head();

    perform_simulation_on_GPU();

    write_heads_to_file();

    free_allocated_memory();

    return 0;
}

void allocate_memory() {
    CUDA_CHECK_RETURN(hipMallocManaged(&(d_read.head), sizeof(double) * ROWS * COLS));
    CUDA_CHECK_RETURN(hipMallocManaged(&(d_write.head), sizeof(double) * ROWS * COLS));
    CUDA_CHECK_RETURN(hipMallocManaged(&(d_read.Sy), sizeof(double) * ROWS * COLS));
    CUDA_CHECK_RETURN(hipMallocManaged(&(d_read.K), sizeof(double) * ROWS * COLS));
    CUDA_CHECK_RETURN(hipMallocManaged(&(d_read.Source), sizeof(double) * ROWS * COLS));
}

void init_read_ca() {
    for (int i = 0; i < ROWS; i++)
        for (int j = 0; j < COLS; j++) {
            d_read.head[i * ROWS + j] = headFixed;
            if (j == COLS - 1) {
                d_read.head[i * ROWS + j] = headCalculated;
            }
            d_read.Sy[i * ROWS + j] = Syinitial;
            d_read.K[i * ROWS + j] = Kinitial;
            d_read.Source[i * ROWS + j] = 0;
        }

    d_read.Source[posSy * ROWS + posSx] = qw;
}

void init_write_head(){
	memcpy(d_write.head, d_read.head, sizeof(double)*ROWS*COLS);
}


void perform_simulation_on_GPU() {
    dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
    for (int i = 0; i < SIMULATION_ITERATIONS; i++) {
    	find_active_cells();
    	int* active_cells_array = thrust::raw_pointer_cast( &d_active_cells_vector[0] );
    	int size = d_active_cells_vector.size();
        const int blockCount = size * size / (BLOCK_SIZE * BLOCK_SIZE);
        double gridSize = sqrt(blockCount) + 1;
        dim3 blockCount2D(gridSize, gridSize);
        simulation_step_kernel << < blockCount2D, blockSize >> > (d_read, d_write.head, active_cells_array, size );

        hipDeviceSynchronize();

        double *tmp1 = d_write.head;
        d_write.head = d_read.head;
        d_read.head = tmp1;
    }
}

void write_heads_to_file() {
    FILE *fp;
    fp = fopen("heads_ca.txt", "w");

    for (int i = 0; i < ROWS; i++) {
        for (int j = 0; j < COLS; j++) {
            fprintf(fp, "%lf, ", d_write.head[i * ROWS + j]);
        }
        fprintf(fp, "\n");
    }

    fclose(fp);
}

void free_allocated_memory(){
	hipFree(d_read.head);
	hipFree(d_write.head);
	hipFree(d_read.Sy);
	hipFree(d_read.K);
	hipFree(d_read.Source);
}

/**
 * Check the return value of the CUDA runtime API call and exit
 * the application if the call has failed.
 */
static void CheckCudaErrorAux(const char *file, unsigned line, const char *statement, hipError_t err) {
    if (err == hipSuccess)
        return;
    std::cerr << statement << " returned " << hipGetErrorString(err) << "(" << err << ") at " << file << ":" << line
              << std::endl;
    exit(1);
}

