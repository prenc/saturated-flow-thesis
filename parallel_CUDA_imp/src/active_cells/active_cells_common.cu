#include "../params.h"
#include "../common/cuda_error_check.cu"
#include "../common/write_heads_to_file.c"

struct CA d_read, d_write;

void allocate_memory() {
	CUDASAFECALL(hipMallocManaged(&(d_read.head), sizeof(double) * ROWS * COLS));
	CUDASAFECALL(hipMallocManaged(&(d_write.head), sizeof(double) * ROWS * COLS));
	CUDASAFECALL(hipMallocManaged(&(d_read.Sy), sizeof(double) * ROWS * COLS));
	CUDASAFECALL(hipMallocManaged(&(d_read.K), sizeof(double) * ROWS * COLS));
	CUDASAFECALL(hipMallocManaged(&(d_read.Source), sizeof(double) * ROWS * COLS));
}

void init_read_ca() {
    for (int i = 0; i < ROWS; i++)
        for (int j = 0; j < COLS; j++) {
            d_read.head[i * ROWS + j] = headFixed;
            if (j == COLS - 1) {
                d_read.head[i * ROWS + j] = headCalculated;
            }
            d_read.Sy[i * ROWS + j] = Syinitial;
            d_read.K[i * ROWS + j] = Kinitial;
            d_read.Source[i * ROWS + j] = 0;
        }

    d_read.Source[posSy * ROWS + posSx] = qw;
}

void init_write_head(){
	memcpy(d_write.head, d_read.head, sizeof(double)*ROWS*COLS);
}

void free_allocated_memory(){
	hipFree(d_read.head);
	hipFree(d_write.head);
	hipFree(d_read.Sy);
	hipFree(d_read.K);
	hipFree(d_read.Source);
}
