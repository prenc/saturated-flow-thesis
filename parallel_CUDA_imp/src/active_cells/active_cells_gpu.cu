#include "hip/hip_runtime.h"
#include "active_cells_common.cu"

#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/unique.h>
#include <thrust/execution_policy.h>

__device__ int active_cells_idx[ROWS*COLS];
__managed__ int dev_count = 0;

__device__ int my_push_back(int mt) {
  int insert_pt = atomicAdd(&dev_count, 1);
  if (insert_pt < ROWS*COLS){
	 active_cells_idx[insert_pt] = mt;
    return insert_pt;
  }
  else return -1;
}

__global__ void simulation_step_kernel(struct CA d_ca, double *d_write_head) {
    unsigned ac_idx_x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned ac_idx_y = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned ac_idx_g = ac_idx_y * COLS + ac_idx_x;

    if(ac_idx_g < dev_count){
        unsigned idx_g = active_cells_idx[ac_idx_g];
        unsigned idx_x = idx_g % COLS;
        unsigned idx_y = idx_g / COLS;
		if (idx_y != 0 && idx_y != ROWS - 1) {
				double Q = 0;
				double diff_head;
				double tmp_t;

				if (idx_x >= 1) {
					diff_head = d_ca.head[idx_g - 1] - d_ca.head[idx_g];
					tmp_t = d_ca.K[idx_g] * THICKNESS;
					Q += diff_head * tmp_t;
				}
				if (idx_y >= 1) {
					diff_head = d_ca.head[(idx_y - 1) * COLS + idx_x] - d_ca.head[idx_g];
					tmp_t = d_ca.K[idx_g] * THICKNESS;
					Q += diff_head * tmp_t;
				}
				if (idx_x + 1 < COLS) {
					diff_head = d_ca.head[idx_g + 1] - d_ca.head[idx_g];
					tmp_t = d_ca.K[idx_g] * THICKNESS;
					Q += diff_head * tmp_t;
				}
				if (idx_y + 1 < ROWS) {
					diff_head = d_ca.head[(idx_y + 1) * COLS + idx_x] - d_ca.head[idx_g];
					tmp_t = d_ca.K[idx_g] * THICKNESS;
					Q += diff_head * tmp_t;
				}

				Q -= d_ca.Source[idx_g];

				double ht1 = Q * DELTA_T;
				double ht2 = AREA * d_ca.Sy[idx_g];

			  d_write_head[idx_g] = d_ca.head[idx_g] + ht1 / ht2;
		}
    }
}
__global__ void find_active_cells_kernel(struct CA d_ca) {
    unsigned idx_x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned idx_y = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned idx_g = idx_y * COLS + idx_x;

    if(idx_x < ROWS && idx_y < COLS ){
		if(d_ca.head[idx_g]  < headFixed || d_ca.Source[idx_g] != 0 ){
			my_push_back(idx_g);
			return;
		}
		if (idx_x >= 1) {
			if(d_ca.head[idx_g - 1] < headFixed){
				my_push_back(idx_g);
				return;
			}
		}
		if (idx_y >= 1) {
			if(d_ca.head[idx_g - COLS] < headFixed){
				my_push_back(idx_g);
				return;
			}
		}
		if (idx_x + 1 < COLS) {
			if(d_ca.head[idx_g + 1] < headFixed){
				my_push_back(idx_g);
				return;
			}
		}
		if (idx_y + 1 < ROWS) {
			if(d_ca.head[idx_g + COLS] < headFixed){
				my_push_back(idx_g);
				return;
			}
		}
    }
}

void perform_simulation_on_GPU() {
	dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
	const int blockCount = (ROWS * COLS) / (BLOCK_SIZE * BLOCK_SIZE) + 1;
	double gridSize = sqrt(blockCount) + 1;
	dim3 blockCount2D(gridSize, gridSize);
	double activeBlockCount, activeGridSize;
	for (int i = 0; i < SIMULATION_ITERATIONS; i++) {
		if(dev_count < ROWS*COLS ){
			dev_count = 0;
			find_active_cells_kernel << < blockCount2D, blockSize >> > (d_read);
			hipDeviceSynchronize();
		}
		activeBlockCount = dev_count* dev_count/ (BLOCK_SIZE * BLOCK_SIZE);
		activeGridSize = sqrt(activeBlockCount) + 1;
		dim3 activeBlockCount2D(activeGridSize, activeGridSize);
		simulation_step_kernel << < activeBlockCount2D, blockSize >> > (d_read, d_write.head);
		hipDeviceSynchronize();

		double *tmp1 = d_write.head;
		d_write.head = d_read.head;
		d_read.head = tmp1;
    }
}

int main(void) {
    allocate_memory();
    init_read_ca();
    init_write_head();

    perform_simulation_on_GPU();

    write_heads_to_file(d_write.head);
    free_allocated_memory();
    return 0;
}









