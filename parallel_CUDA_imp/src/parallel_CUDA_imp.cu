#include "hip/hip_runtime.h"
/*
 ============================================================================
 Name        : parallel_CUDA_imp.cu
 Author      : Tomasz Pęcak and Paweł Renc
 Version     :
 Copyright   : Your copyright notice
 Description : CUDA compute reciprocals
 ============================================================================
 */

#include <iostream>
#include <numeric>
#include <stdlib.h>
#include <stdio.h>

//MODEL PARAMS

#define ROWS 100
#define COLS 100
#define CELL_SIZE_X 10
#define CELL_SIZE_Y 10
#define THICKNESS 50

#define Syinitial 0.1
#define Kinitial  0.0000125

#define headFixed 50
#define headCalculated 50

#define SIMULATION_ITERATIONS 1000
#define BLOCK_SIZE 256

double delta_t_ = 4000;
double qw = 0.001;

int posSy = ROWS / 2;
int posSx = COLS / 2;

struct CA {
    double *head;
    double *Sy;
    double *K;
    double *Source;
} h_ca, d_read, d_write;

double *d_write_head;

void init_host_ca();

static void CheckCudaErrorAux(const char *, unsigned, const char *, hipError_t);

#define CUDA_CHECK_RETURN(value) CheckCudaErrorAux(__FILE__,__LINE__, #value, value)

/**
 * CUDA kernel that computes simulation step
 */
__global__ void simulation_step_kernel(CA data, double *d_write_head) {
    unsigned idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < ROWS * COLS) {
        d_write_head[idx] = data.head[idx] * 2;
    }
}

/**
 * Host function that copies the data
 */
void copy_data_from_CPU_to_GPU() {
    double *d_read_head, *d_read_Sy, *d_read_K, *d_read_Source;
    CUDA_CHECK_RETURN(hipMalloc(&d_read_head, sizeof(double) * ROWS * COLS));
    CUDA_CHECK_RETURN(hipMalloc(&d_write_head, sizeof(double) * ROWS * COLS));
    CUDA_CHECK_RETURN(hipMalloc(&d_read_Sy, sizeof(double) * ROWS * COLS));
    CUDA_CHECK_RETURN(hipMalloc(&d_read_K, sizeof(double) * ROWS * COLS));
    CUDA_CHECK_RETURN(hipMalloc(&d_read_Source, sizeof(double) * ROWS * COLS));

    CUDA_CHECK_RETURN(hipMemcpy(d_read_head, h_ca.head, sizeof(double) * ROWS * COLS, hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(d_write_head, h_ca.head, sizeof(double) * ROWS * COLS, hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(d_read_Sy, h_ca.Sy, sizeof(double) * ROWS * COLS, hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(d_read_K, h_ca.K, sizeof(double) * ROWS * COLS, hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(d_read_Source, h_ca.Source, sizeof(double) * ROWS * COLS, hipMemcpyHostToDevice));
}

void copy_data_from_GPU_to_CPU() {
    CUDA_CHECK_RETURN(hipMemcpy(h_ca.head, d_write_head, sizeof(double) * ROWS * COLS, hipMemcpyDeviceToHost));
}

void perform_simulation_on_GPU() {
    const int blockCount = (ROWS * COLS) / BLOCK_SIZE + 1;
    for (int i = 0; i < SIMULATION_ITERATIONS; i++) {
        simulation_step_kernel << < blockCount, BLOCK_SIZE >> > (d_read, d_write_head);
    }
}

int main(void) {
    h_ca.head = new double[ROWS * COLS]();
    h_ca.Sy = new double[ROWS * COLS]();
    h_ca.K = new double[ROWS * COLS]();
    h_ca.Source = new double[ROWS * COLS]();

    init_host_ca();

    copy_data_from_CPU_to_GPU();

    printf("%lf", h_ca.head[100]);
    perform_simulation_on_GPU();

    copy_data_from_GPU_to_CPU();
    printf("%lf", h_ca.head[100]);
    return 0;
}

void init_host_ca() {
    for (int i = 0; i < ROWS; i++)
        for (int j = 0; j < COLS; j++) {
            h_ca.head[i * ROWS + j] = headFixed;
            if (j == COLS - 1) {
                h_ca.head[i * ROWS + j] = headCalculated;
            }
            h_ca.Sy[i * ROWS + j] = Syinitial;
            h_ca.K[i * ROWS + j] = Kinitial;
            h_ca.Source[i * ROWS + j] = 0;
        }

    h_ca.Source[posSy * ROWS + posSx] = qw;
}

/**
 * Check the return value of the CUDA runtime API call and exit
 * the application if the call has failed.
 */
static void CheckCudaErrorAux(const char *file, unsigned line, const char *statement, hipError_t err) {
    if (err == hipSuccess)
        return;
    std::cerr << statement << " returned " << hipGetErrorString(err) << "(" << err << ") at " << file << ":" << line
              << std::endl;
    exit(1);
}

