#include "hip/hip_runtime.h"
#include "global_memory_common.h"

__global__ void simulation_step_kernel(struct CA *d_ca, double *d_write_head) {
    unsigned idx_x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned idx_y = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned idx_g = idx_y * COLS + idx_x;

    double Q, diff_head, tmp_t, ht1, ht2;
    if (idx_x < COLS && idx_y < ROWS)
        if (idx_y != 0 && idx_y != ROWS - 1) {
            Q = 0;
        	if (idx_x >= 1) {
                diff_head = d_ca->head[idx_g - 1] - d_ca->head[idx_g];
                tmp_t = d_ca->K[idx_g] * THICKNESS;
                Q += diff_head * tmp_t;
            }
            if (idx_y >= 1) {
                diff_head = d_ca->head[(idx_y - 1) * COLS + idx_x] - d_ca->head[idx_g];
                tmp_t = d_ca->K[idx_g] * THICKNESS;
                Q += diff_head * tmp_t;
            }
            if (idx_x + 1 < COLS) {
                diff_head = d_ca->head[idx_g + 1] - d_ca->head[idx_g];
                tmp_t = d_ca->K[idx_g] * THICKNESS;
                Q += diff_head * tmp_t;
            }
            if (idx_y + 1 < ROWS) {
                diff_head = d_ca->head[(idx_y + 1) * COLS + idx_x] - d_ca->head[idx_g];
                tmp_t = d_ca->K[idx_g] * THICKNESS;
                Q += diff_head * tmp_t;
            }

            Q -= d_ca->Source[idx_g];

            ht1 = Q * DELTA_T;
            ht2 = AREA * d_ca->Sy[idx_g];

          d_write_head[idx_g] = d_ca->head[idx_g] + ht1 / ht2;
        }
}

void perform_simulation_on_GPU() {
    dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
    const int blockCount = (ROWS * COLS) / (BLOCK_SIZE * BLOCK_SIZE) + 1;
    double gridSize = sqrt(blockCount) + 1;
    dim3 blockCount2D(gridSize, gridSize);
    for (int i = 0; i < SIMULATION_ITERATIONS; i++) {
        simulation_step_kernel << < blockCount2D, blockSize >> > (d_read_ca, d_write_head);

        hipDeviceSynchronize();

        double *tmp1 = d_write_head;
        CUDASAFECALL(
                hipMemcpy(&d_write_head, &(d_read_ca->head), sizeof(d_read_ca->head), hipMemcpyDeviceToHost));
        CUDASAFECALL(hipMemcpy(&(d_read_ca->head), &tmp1, sizeof(tmp1), hipMemcpyHostToDevice));
    }
}

int main(void) {
    init_host_ca();
    copy_data_from_CPU_to_GPU();

    perform_simulation_on_GPU();

    return 0;
}
