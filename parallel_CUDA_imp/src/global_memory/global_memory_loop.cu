#include "hip/hip_runtime.h"
#include <iostream>
#include <numeric>
#include <stdlib.h>
#include <stdio.h>
#include <math.h>

//MODEL PARAMS

#define ROWS 1000
#define COLS 1000

#define CELL_SIZE_X 10
#define CELL_SIZE_Y 10
#define AREA CELL_SIZE_X*CELL_SIZE_Y

#define THICKNESS 50

#define Syinitial 0.1
#define Kinitial  0.0000125

#define headFixed 50
#define headCalculated 50

#define SIMULATION_ITERATIONS 1000
#define BLOCK_SIZE 16

#define KERNEL_LOOP_SIZE 100

#define DELTA_T 4000;
double qw = 0.001;

int posSy = ROWS / 2;
int posSx = COLS / 2;

struct CA {
    double *head;
    double *Sy;
    double *K;
    double *Source;
} h_ca, d_read, d_write;

double *d_write_head;
CA *d_read_ca;

void init_host_ca();

static void CheckCudaErrorAux(const char *, unsigned, const char *, hipError_t);

#define CUDA_CHECK_RETURN(value) CheckCudaErrorAux(__FILE__,__LINE__, #value, value)

__global__ void simulation_step_kernel(struct CA *d_ca, double *d_write_head) {
    unsigned idx_x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned idx_y = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned idx_g = idx_y * COLS + idx_x;

    double Q = 0, diff_head,tmp_t, ht1, ht2;
    for(int i = 0; i < KERNEL_LOOP_SIZE; i++){
    	 if (idx_x < COLS && idx_y < ROWS)
    	        if (idx_y != 0 && idx_y != ROWS - 1) {


    	            if (idx_x >= 1) {
    	                diff_head = d_ca->head[idx_g - 1] - d_ca->head[idx_g];
    	                tmp_t = d_ca->K[idx_g] * THICKNESS;
    	                Q += diff_head * tmp_t;
    	            }
    	            if (idx_y >= 1) {
    	                diff_head = d_ca->head[(idx_y - 1) * COLS + idx_x] - d_ca->head[idx_g];
    	                tmp_t = d_ca->K[idx_g] * THICKNESS;
    	                Q += diff_head * tmp_t;
    	            }
    	            if (idx_x + 1 < COLS) {
    	                diff_head = d_ca->head[idx_g + 1] - d_ca->head[idx_g];
    	                tmp_t = d_ca->K[idx_g] * THICKNESS;
    	                Q += diff_head * tmp_t;
    	            }
    	            if (idx_y + 1 < ROWS) {
    	                diff_head = d_ca->head[(idx_y + 1) * COLS + idx_x] - d_ca->head[idx_g];
    	                tmp_t = d_ca->K[idx_g] * THICKNESS;
    	                Q += diff_head * tmp_t;
    	            }

    	            Q -= d_ca->Source[idx_g];

    	            ht1 = Q * DELTA_T;
    	            ht2 = AREA * d_ca->Sy[idx_g];

    	          d_write_head[idx_g] = d_ca->head[idx_g] + ht1 / ht2;
    	        }
    }
}

void copy_data_from_CPU_to_GPU() {
    double *d_read_head, *d_read_Sy, *d_read_K, *d_read_Source;

    CUDA_CHECK_RETURN(hipMalloc((void **) &d_read_ca, sizeof(*d_read_ca)));
    CUDA_CHECK_RETURN(hipMalloc((void **) &d_read_head, sizeof(*d_read_head) * ROWS * COLS));
    CUDA_CHECK_RETURN(hipMalloc((void **) &d_write_head, sizeof(double) * ROWS * COLS));
    CUDA_CHECK_RETURN(hipMalloc(&d_read_Sy, sizeof(double) * ROWS * COLS));
    CUDA_CHECK_RETURN(hipMalloc(&d_read_K, sizeof(double) * ROWS * COLS));
    CUDA_CHECK_RETURN(hipMalloc(&d_read_Source, sizeof(double) * ROWS * COLS));

    CUDA_CHECK_RETURN(hipMemcpy(d_read_head, h_ca.head, sizeof(*d_read_head) * ROWS * COLS, hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(&(d_read_ca->head), &d_read_head, sizeof(d_read_ca->head), hipMemcpyHostToDevice));

    CUDA_CHECK_RETURN(hipMemcpy(d_read_Sy, h_ca.Sy, sizeof(double) * ROWS * COLS, hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(&(d_read_ca->Sy), &d_read_Sy, sizeof(d_read_ca->Sy), hipMemcpyHostToDevice));

    CUDA_CHECK_RETURN(hipMemcpy(d_read_K, h_ca.K, sizeof(double) * ROWS * COLS, hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(&(d_read_ca->K), &d_read_K, sizeof(d_read_ca->K), hipMemcpyHostToDevice));

    CUDA_CHECK_RETURN(hipMemcpy(d_read_Source, h_ca.Source, sizeof(double) * ROWS * COLS, hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(
            hipMemcpy(&(d_read_ca->Source), &d_read_Source, sizeof(d_read_ca->Source), hipMemcpyHostToDevice));

    CUDA_CHECK_RETURN(hipMemcpy(d_write_head, h_ca.head, sizeof(double) * ROWS * COLS, hipMemcpyHostToDevice));

}

void copy_data_from_GPU_to_CPU() {
    CUDA_CHECK_RETURN(hipMemcpy(h_ca.head, d_write_head, sizeof(double) * ROWS * COLS, hipMemcpyDeviceToHost));
}

void perform_simulation_on_GPU() {

    dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
    const int blockCount = (ROWS * COLS) / (BLOCK_SIZE * BLOCK_SIZE) + 1;
    double gridSize = sqrt(blockCount) + 1;
    dim3 blockCount2D(gridSize, gridSize);
    for (int i = 0; i < SIMULATION_ITERATIONS; i++) {
        simulation_step_kernel << < blockCount2D, blockSize >> > (d_read_ca, d_write_head);

        hipDeviceSynchronize();

        double *tmp1 = d_write_head;
        CUDA_CHECK_RETURN(
                hipMemcpy(&d_write_head, &(d_read_ca->head), sizeof(d_read_ca->head), hipMemcpyDeviceToHost));
        CUDA_CHECK_RETURN(hipMemcpy(&(d_read_ca->head), &tmp1, sizeof(tmp1), hipMemcpyHostToDevice));
    }
}

void write_heads_to_file() {
    FILE *fp;
    fp = fopen("heads_ca.txt", "w");

    for (int i = 0; i < ROWS; i++) {
        for (int j = 0; j < COLS; j++) {
            fprintf(fp, "%lf, ", h_ca.head[i * ROWS + j]);
        }
        fprintf(fp, "\n");
    }

    fclose(fp);
}

int main(void) {
    init_host_ca();

    copy_data_from_CPU_to_GPU();

    perform_simulation_on_GPU();

    copy_data_from_GPU_to_CPU();

    write_heads_to_file();

    return 0;
}

void init_host_ca() {
    h_ca.head = new double[ROWS * COLS]();
    h_ca.Sy = new double[ROWS * COLS]();
    h_ca.K = new double[ROWS * COLS]();
    h_ca.Source = new double[ROWS * COLS]();

    for (int i = 0; i < ROWS; i++)
        for (int j = 0; j < COLS; j++) {
            h_ca.head[i * ROWS + j] = headFixed;
            if (j == COLS - 1) {
                h_ca.head[i * ROWS + j] = headCalculated;
            }
            h_ca.Sy[i * ROWS + j] = Syinitial;
            h_ca.K[i * ROWS + j] = Kinitial;
            h_ca.Source[i * ROWS + j] = 0;
        }

    h_ca.Source[posSy * ROWS + posSx] = qw;
}

/**
 * Check the return value of the CUDA runtime API call and exit
 * the application if the call has failed.
 */
static void CheckCudaErrorAux(const char *file, unsigned line, const char *statement, hipError_t err) {
    if (err == hipSuccess)
        return;
    std::cerr << statement << " returned " << hipGetErrorString(err) << "(" << err << ") at " << file << ":" << line
              << std::endl;
    exit(1);
}

