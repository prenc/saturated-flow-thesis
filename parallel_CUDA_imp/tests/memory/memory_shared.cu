#include "hip/hip_runtime.h"
#include "params.h"

struct CA {
    double *head;
    double *Sy;
    double *K;
    double *Source;
} h_ca;

double *d_write_head;
CA *d_read_ca;

void init_host_ca();

static void CheckCudaErrorAux(const char *, unsigned, const char *, hipError_t);

#define CUDA_CHECK_RETURN(value) CheckCudaErrorAux(__FILE__,__LINE__, #value, value)

__global__ void simulation_step_kernel(struct CA *d_ca, double *d_write_head, int grid_size) {
    __shared__ double s_heads[BLOCK_SIZE + 2][BLOCK_SIZE + 2];
    __shared__ double s_K[BLOCK_SIZE + 2][BLOCK_SIZE + 2];
    unsigned idx_x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned idx_y = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned idx_g = idx_y * COLS + idx_x;

    unsigned x = threadIdx.x + 1;
    unsigned y = threadIdx.y + 1;

    s_heads[y][x] = d_ca->head[idx_g];
    s_K[y][x] = d_ca->K[idx_g];

    if (threadIdx.x == 0 && blockIdx.x != 0) // left
        s_heads[y][x - 1] = d_ca->head[idx_g - 1];
    if (threadIdx.x == BLOCK_SIZE - 1 && blockIdx.x != grid_size - 1) // right
        s_heads[y][x + 1] = d_ca->head[idx_g + 1];
    if (threadIdx.y == 0 && blockIdx.y != 0) // upper
        s_heads[y - 1][x] = d_ca->head[idx_g - COLS];
    if (threadIdx.y == BLOCK_SIZE - 1 && blockIdx.y != grid_size - 1) // bottom
        s_heads[y + 1][x] = d_ca->head[idx_g + COLS];

    __syncthreads();

    double Q = 0;
    double diff_head;
    double tmp_t;

    if (idx_x < COLS && idx_y < ROWS)
        if (idx_y != 0 && idx_y != ROWS - 1) {
            if (idx_x >= 1) { // left neighbor
                diff_head = s_heads[y][x - 1] - s_heads[y][x];
                tmp_t = s_K[y][x] * THICKNESS;
                Q += diff_head * tmp_t;
            }
            if (idx_y >= 1) { // upper neighbor
                diff_head = s_heads[y - 1][x] - s_heads[y][x];
                tmp_t = s_K[y][x] * THICKNESS;
                Q += diff_head * tmp_t;
            }
            if (idx_x + 1 < COLS) { // right neighbor
                diff_head = s_heads[y][x + 1] - s_heads[y][x];
                tmp_t = s_K[y][x] * THICKNESS;
                Q += diff_head * tmp_t;
            }
            if (idx_y + 1 < ROWS) { // bottom neighbor
                diff_head = s_heads[y + 1][x] - s_heads[y][x];
                tmp_t = s_K[y][x] * THICKNESS;
                Q += diff_head * tmp_t;
            }

            Q -= d_ca->Source[idx_g];

            double ht1 = Q * DELTA_T;
            double ht2 = AREA * d_ca->Sy[idx_g];

            d_write_head[idx_g] = s_heads[y][x] + ht1 / ht2;
        }
}

void copy_data_from_CPU_to_GPU() {
    double *d_head, *d_Sy, *d_K, *d_Source;

    CUDA_CHECK_RETURN(hipMalloc(&d_read_ca, sizeof(*d_read_ca)));
    CUDA_CHECK_RETURN(hipMalloc(&d_head, sizeof(*d_head) * ROWS * COLS));
    CUDA_CHECK_RETURN(hipMalloc(&d_write_head, sizeof(double) * ROWS * COLS));
    CUDA_CHECK_RETURN(hipMalloc(&d_Sy, sizeof(double) * ROWS * COLS));
    CUDA_CHECK_RETURN(hipMalloc(&d_K, sizeof(double) * ROWS * COLS));
    CUDA_CHECK_RETURN(hipMalloc(&d_Source, sizeof(double) * ROWS * COLS));

    CUDA_CHECK_RETURN(hipMemcpy(d_head, h_ca.head, sizeof(*d_head) * ROWS * COLS, hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(&(d_read_ca->head), &d_head, sizeof(d_read_ca->head), hipMemcpyHostToDevice));

    CUDA_CHECK_RETURN(hipMemcpy(d_Sy, h_ca.Sy, sizeof(double) * ROWS * COLS, hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(&(d_read_ca->Sy), &d_Sy, sizeof(d_read_ca->Sy), hipMemcpyHostToDevice));

    CUDA_CHECK_RETURN(hipMemcpy(d_K, h_ca.K, sizeof(double) * ROWS * COLS, hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(&(d_read_ca->K), &d_K, sizeof(d_read_ca->K), hipMemcpyHostToDevice));

    CUDA_CHECK_RETURN(hipMemcpy(d_Source, h_ca.Source, sizeof(double) * ROWS * COLS, hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(
            hipMemcpy(&(d_read_ca->Source), &d_Source, sizeof(d_read_ca->Source), hipMemcpyHostToDevice));

    CUDA_CHECK_RETURN(hipMemcpy(d_write_head, h_ca.head, sizeof(double) * ROWS * COLS, hipMemcpyHostToDevice));
}

void copy_data_from_GPU_to_CPU() {
    CUDA_CHECK_RETURN(hipMemcpy(h_ca.head, d_write_head, sizeof(double) * ROWS * COLS, hipMemcpyDeviceToHost));
}

void perform_simulation_on_GPU() {
    dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE);
    const int blockCount = ceil((ROWS * COLS) / (BLOCK_SIZE * BLOCK_SIZE));
    double gridSize = ceil(sqrt(blockCount));
    dim3 gridDim(gridSize, gridSize);

    for (int i = 0; i < SIMULATION_ITERATIONS; i++) {
        simulation_step_kernel << < gridDim, blockDim >> > (d_read_ca, d_write_head, gridSize);

        hipDeviceSynchronize();

        double *tmp1 = d_write_head;
        CUDA_CHECK_RETURN(
                hipMemcpy(&d_write_head, &(d_read_ca->head), sizeof(d_read_ca->head), hipMemcpyDeviceToHost));
        CUDA_CHECK_RETURN(hipMemcpy(&(d_read_ca->head), &tmp1, sizeof(tmp1), hipMemcpyHostToDevice));
    }
}

void write_heads_to_file() {
    FILE *fp;
    fp = fopen("memory_shared_heads.txt", "w");

    for (int i = 0; i < ROWS; i++) {
        for (int j = 0; j < COLS; j++) {
            fprintf(fp, "%lf, ", h_ca.head[i * ROWS + j]);
        }
        fprintf(fp, "\n");
    }

    fclose(fp);
}

int main(void) {
    init_host_ca();

    copy_data_from_CPU_to_GPU();

    perform_simulation_on_GPU();

    copy_data_from_GPU_to_CPU();

    write_heads_to_file();

    return 0;
}

void init_host_ca() {
    h_ca.head = new double[ROWS * COLS]();
    h_ca.Sy = new double[ROWS * COLS]();
    h_ca.K = new double[ROWS * COLS]();
    h_ca.Source = new double[ROWS * COLS]();

    for (int i = 0; i < ROWS; i++)
        for (int j = 0; j < COLS; j++) {
            h_ca.head[i * ROWS + j] = headFixed;
            if (j == COLS - 1) {
                h_ca.head[i * ROWS + j] = headCalculated;
            }
            h_ca.Sy[i * ROWS + j] = Syinitial;
            h_ca.K[i * ROWS + j] = Kinitial;
            h_ca.Source[i * ROWS + j] = 0;
        }

    h_ca.Source[posSy * ROWS + posSx] = qw;
}

/**
 * Check the return value of the CUDA runtime API call and exit
 * the application if the call has failed.
 */
static void CheckCudaErrorAux(const char *file, unsigned line, const char *statement, hipError_t err) {
    if (err == hipSuccess)
        return;
    std::cerr << statement << " returned " << hipGetErrorString(err) << "(" << err << ") at " << file << ":" << line
              << std::endl;
    exit(1);
}

