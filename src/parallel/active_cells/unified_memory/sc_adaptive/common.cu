#include <thrust/device_vector.h>
#include "../../../common/memory_management.cuh"
#include "../../../common/statistics.h"
#include "../../../kernels/transition_kernels.cu"
#include "../../../kernels/dummy_kernels.cu"
#include "../../../kernels/ac_kernels.cu"
#include "../../utils.cu"


int main(int argc, char *argv[])
{
    auto h_ca = new CA();
    double *headsWrite;
    allocateManagedMemory(h_ca, headsWrite);

    initializeCA(h_ca);
    memcpy(headsWrite, h_ca->heads, sizeof(double) * ROWS * COLS);

    thrust::device_vector<int> activeCellsMask(ROWS * COLS, -1);
    thrust::device_vector<int> activeCellsIds(ROWS * COLS, -1);

    ac_utils::mark_sources_as_active_cells(h_ca, thrust::raw_pointer_cast(&activeCellsIds[0]));

    dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridDims = calculate_grid_dim();

    std::vector<StatPoint> stats;
    Timer stepTimer, activeCellsEvalTimer, transitionTimer;

    auto standardIterationTime = ac_utils::measure_standard_iteration_time(h_ca, headsWrite);

    bool isWholeGridActive = false;
    int devActiveCellsCount;
    int acIterCounter{};
    stepTimer.start();
    for (int i{}; i < SIMULATION_ITERATIONS; ++i)
    {
        if (!isWholeGridActive)
        {
            activeCellsEvalTimer.start();
            thrust::copy_if(thrust::device, activeCellsMask.begin(), activeCellsMask.end(),
                            activeCellsIds.begin(), is_not_minus_one<int>());
            devActiveCellsCount = thrust::count_if(activeCellsIds.begin(), activeCellsIds.end(),
                                                   is_not_minus_one<int>());
            activeCellsEvalTimer.stop();

            isWholeGridActive = devActiveCellsCount == ROWS * COLS;

            dim3 activeGridDim = calculate_grid_dim(devActiveCellsCount);

            transitionTimer.start();
            ac_kernels::sc <<< activeGridDim, blockSize >>>(
                    *h_ca, headsWrite, thrust::raw_pointer_cast(&activeCellsIds[0]),
                    thrust::raw_pointer_cast(&activeCellsMask[0]),
                    devActiveCellsCount);

            for (int j = 0; j < EXTRA_KERNELS; j++)
            {
                dummy_kernels::dummy_active_sc <<< activeGridDim, blockSize >>>(
                        *h_ca, headsWrite, thrust::raw_pointer_cast(&activeCellsIds[0]),
                        devActiveCellsCount);
            }
            if (acIterCounter > 5)
            {
                isWholeGridActive = true;
                devActiveCellsCount = ROWS * COLS;
                activeCellsEvalTimer.start();
                activeCellsEvalTimer.stop();
            }
        }
        else
        {
            transitionTimer.start();
            kernels::standard_step <<< gridDims, blockSize >>>(*h_ca, headsWrite);

            for (int j = 0; j < EXTRA_KERNELS; j++)
            {
                dummy_kernels::dummy_active_naive <<< gridDims, blockSize >>>(*h_ca, headsWrite);
            }
        }

        ERROR_CHECK(hipDeviceSynchronize());
        transitionTimer.stop();

        std::swap(h_ca->heads, headsWrite);

        if (i % STATISTICS_WRITE_FREQ == STATISTICS_WRITE_FREQ - 1)
        {
            stepTimer.stop();
            auto stat = new StatPoint(
                    devActiveCellsCount / (double) (ROWS * COLS),
                    stepTimer.elapsedNanoseconds(),
                    transitionTimer.elapsedNanoseconds(),
                    activeCellsEvalTimer.elapsedNanoseconds());
            stat->adaptiveTime = standardIterationTime;
            if (stepTimer.elapsedNanoseconds() / STATISTICS_WRITE_FREQ >= standardIterationTime)
            {
                acIterCounter++;
            }
            else
            {
                acIterCounter = 0;
            }
            stats.push_back(*stat);
            stepTimer.start();
        }
    }

    if (WRITE_OUTPUT_TO_FILE)
    {
        saveHeadsInFile(h_ca->heads, argv[0]);
    }

    if (WRITE_STATISTICS_TO_FILE)
    {
        writeStatisticsToFile(stats, argv[0]);
    }

    freeAllocatedMemory(h_ca, headsWrite);
}
