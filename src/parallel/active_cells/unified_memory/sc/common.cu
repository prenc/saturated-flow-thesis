#include <thrust/device_vector.h>

#include "../../../common/memory_management.cuh"
#include "../../../common/statistics.h"
#include "../../../kernels/transition_kernels.cu"
#include "../../../kernels/dummy_kernels.cu"
#include "../../../kernels/ac_kernels.cu"
#include "../../utils.cu"

int main(int argc, char *argv[])
{
    auto h_ca = new CA();
    double *headsWrite;
    allocateManagedMemory(h_ca, headsWrite);
    initializeCA(h_ca);

    thrust::device_vector<int> activeCellsMask(ROWS * COLS, -1);
    thrust::device_vector<int> activeCellsIds(ROWS * COLS, -1);

    for (size_t i{0}; i < ROWS * COLS; ++i)
    {
        if (h_ca->sources[i] != 0)
        {
            activeCellsMask[i] = i;
        }
    }

    dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridDims = calculate_grid_dim();

    std::vector<StatPoint> stats;
    Timer stepTimer;
    stepTimer.start();

    bool isWholeGridActive = false;
    int devActiveCellsCount;

    for (int i{}; i < SIMULATION_ITERATIONS; ++i)
    {
        if (!isWholeGridActive)
        {
            thrust::copy_if(thrust::device, activeCellsMask.begin(), activeCellsMask.end(),
                            activeCellsIds.begin(), is_not_minus_one<int>());
            devActiveCellsCount = thrust::count_if(activeCellsIds.begin(), activeCellsIds.end(),
                                                   is_not_minus_one<int>());

            dim3 activeGridDim = calculate_grid_dim(devActiveCellsCount);

            ac_kernels::sc <<< activeGridDim, blockSize >>>(
			        *h_ca, headsWrite, thrust::raw_pointer_cast(&activeCellsIds[0]),
			        thrust::raw_pointer_cast(&activeCellsMask[0]),
			        devActiveCellsCount);
            for (int j = 0; j < EXTRA_KERNELS; j++)
            {
                dummy_kernels::dummy_active_sc <<< activeGridDim, blockSize >>>(
                            *h_ca,
                            headsWrite,
                            thrust::raw_pointer_cast(&activeCellsIds[0]),
                            devActiveCellsCount
                        );
            }

            isWholeGridActive = devActiveCellsCount == ROWS * COLS;
        }
        else
        {
	        kernels::standard_step <<< gridDims, blockSize >>>(*h_ca, headsWrite);
            for (int j = 0; j < EXTRA_KERNELS; j++)
            {
                dummy_kernels::dummy_active_naive <<< gridDims, blockSize >>>(*h_ca, headsWrite);
            }
        }
        ERROR_CHECK(hipDeviceSynchronize());

        std::swap(h_ca->heads, headsWrite);

        save_step_stats(stats, &stepTimer, i, devActiveCellsCount);
    }
    save_output_and_free_memory(argv, h_ca, headsWrite, stats);
}