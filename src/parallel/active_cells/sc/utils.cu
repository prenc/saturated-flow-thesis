#ifndef AC_UTILS
#define AC_UTILS

#include <thrust/device_vector.h>
#include <algorithm>
#include "../../common/memory_management.cuh"
#include "../../kernels/transition_kernels.cu"
#include "../../common/statistics.h"

namespace ac_utils {
    size_t measure_standard_iteration_time(struct CA *h_ca, double *headsWrite){
        Timer stepTimer;
        std::vector<size_t> times{};
        dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
        dim3 gridDims = calculate_grid_dim();

        for (size_t i{}; i < 5; ++i)
        {
            stepTimer.start();
            kernels::standard_step <<< gridDims, blockSize >>>(*h_ca, headsWrite);
            ERROR_CHECK(hipDeviceSynchronize());
            stepTimer.stop();
            times.push_back(stepTimer.elapsedNanoseconds);
        }
        std::sort(times.begin(), times.end());
        return times[0];
    }

    void set_sc_counter(Timer stepTimer, int stepCounter, int *sc_steps_with_higher_time_than_standard,
                        double standardIterationTime){
        if (stepCounter % STATISTICS_WRITE_FREQ == STATISTICS_WRITE_FREQ - 1)
        {
            if (stepTimer.elapsedNanoseconds / STATISTICS_WRITE_FREQ >= standardIterationTime){
                *sc_steps_with_higher_time_than_standard += 1;
            }else{
                *sc_steps_with_higher_time_than_standard = 0;
            }
        }
    }
    bool check_if_model_should_adapt(int sc_steps_with_higher_time_than_standard, int *devActiveCellsCount){
        if (sc_steps_with_higher_time_than_standard > MINIMUM_SC_RUNS_WITH_HIGHER_TIME_TO_ADAPT)
        {
            *devActiveCellsCount = ROWS * COLS;
            return true;
        }
        return false;
    }

}

#endif
