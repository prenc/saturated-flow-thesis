#include "hip/hip_runtime.h"
#include "../../../common/memory_management.cuh"
#include "../../../common/statistics.h"

__device__ unsigned activeCellsIdx[ROWS * COLS];

__global__ void simulation_step_kernel(struct CA ca, double *headsWrite)
{
    unsigned idx_x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned idx_y = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned idx_g = idx_y * blockDim.x * gridDim.x + idx_x;

    if (idx_g < ROWS * COLS)
    {
        if (activeCellsIdx[idx_g] == 1)
        {
            idx_x = idx_g % COLS;
            idx_y = idx_g / COLS;

            double Q{}, diff_head, tmp_t, ht1, ht2;
#ifdef LOOP
            for (int i = 0; i < KERNEL_LOOP_SIZE; i++)
            {
                if (i == KERNEL_LOOP_SIZE - 1)
                {
                    if (Q) { Q = 0; }
                }
#endif
            if (idx_x >= 1)
            {
                diff_head = ca.heads[idx_g - 1] - ca.heads[idx_g];
                tmp_t = ca.K[idx_g] * THICKNESS;
                Q += diff_head * tmp_t;
            }
            if (idx_y >= 1)
            {
                diff_head = ca.heads[(idx_y - 1) * COLS + idx_x] - ca.heads[idx_g];
                tmp_t = ca.K[idx_g] * THICKNESS;
                Q += diff_head * tmp_t;
            }
            if (idx_x + 1 < COLS)
            {
                diff_head = ca.heads[idx_g + 1] - ca.heads[idx_g];
                tmp_t = ca.K[idx_g] * THICKNESS;
                Q += diff_head * tmp_t;
            }
            if (idx_y + 1 < ROWS)
            {
                diff_head = ca.heads[(idx_y + 1) * COLS + idx_x] - ca.heads[idx_g];
                tmp_t = ca.K[idx_g] * THICKNESS;
                Q += diff_head * tmp_t;
            }
#ifdef LOOP
            }
#endif
            Q -= ca.sources[idx_g];
            ht1 = Q * DELTA_T;
            ht2 = AREA * ca.Sy[idx_g];

            headsWrite[idx_g] = ca.heads[idx_g] + ht1 / ht2;
            if (headsWrite[idx_g] < 0)
            { headsWrite[idx_g] = 0; }
        }
    }
}

__global__ void findActiveCells(struct CA d_ca)
{
    unsigned idx_x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned idx_y = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned idx_g = idx_y * COLS + idx_x;

    if (idx_x < ROWS && idx_y < COLS)
    {
        if (d_ca.heads[idx_g] < INITIAL_HEAD || d_ca.sources[idx_g] != 0)
        {
            activeCellsIdx[idx_g] = 1;
            return;
        }
        if (idx_x > 0)
        {
            if (d_ca.heads[idx_g - 1] < INITIAL_HEAD)
            {
                activeCellsIdx[idx_g] = 1;
                return;
            }
        }
        if (idx_y > 0)
        {
            if (d_ca.heads[idx_g - COLS] < INITIAL_HEAD)
            {
                activeCellsIdx[idx_g] = 1;
                return;
            }
        }
        if (idx_x < COLS - 1)
        {
            if (d_ca.heads[idx_g + 1] < INITIAL_HEAD)
            {
                activeCellsIdx[idx_g] = 1;
                return;
            }
        }
        if (idx_y < ROWS - 1)
        {
            if (d_ca.heads[idx_g + COLS] < INITIAL_HEAD)
            {
                activeCellsIdx[idx_g] = 1;
                return;
            }
        }
        activeCellsIdx[idx_g] = 0;
    }
}

int main(int argc, char *argv[])
{
    CA *d_ca = new CA();
    CA *h_ca = new CA();
    double *headsWrite;

    h_ca->heads = new double[ROWS * COLS]();
    h_ca->Sy = new double[ROWS * COLS]();
    h_ca->K = new double[ROWS * COLS]();
    h_ca->sources = new double[ROWS * COLS]();

    initializeCA(h_ca);

    allocateMemory(d_ca, headsWrite);
    copyDataFromCpuToGpu(h_ca, d_ca, headsWrite);

    dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
    const int blockCount = ceil((double) (ROWS * COLS) / (BLOCK_SIZE * BLOCK_SIZE));
    int gridSize = ceil(sqrt(blockCount));
    dim3 gridDims(gridSize, gridSize);

    std::vector<StatPoint> stats;
    Timer stepTimer, activeCellsEvalTimer, transitionTimer;
    stepTimer.start();

    for (int i{}; i < SIMULATION_ITERATIONS; ++i)
    {
        activeCellsEvalTimer.start();
        findActiveCells <<< gridDims, blockSize >>>(*d_ca);
        ERROR_CHECK(hipDeviceSynchronize());
        activeCellsEvalTimer.stop();

        transitionTimer.start();
        simulation_step_kernel <<< gridDims, blockSize >>>(*d_ca, headsWrite);
        ERROR_CHECK(hipDeviceSynchronize());
        transitionTimer.stop();

        double *tmpHeads = d_ca->heads;
        d_ca->heads = headsWrite;
        headsWrite = tmpHeads;

        if (i % STATISTICS_WRITE_FREQ == STATISTICS_WRITE_FREQ - 1)
        {
            stepTimer.stop();
            auto stat = new StatPoint(
                    -1,
                    stepTimer.elapsedNanoseconds(),
                    transitionTimer.elapsedNanoseconds(),
                    activeCellsEvalTimer.elapsedNanoseconds());
            stats.push_back(*stat);
            stepTimer.start();
        }
    }

    if (WRITE_OUTPUT_TO_FILE)
    {
        copyDataFromGpuToCpu(h_ca, d_ca);
        saveHeadsInFile(h_ca->heads, argv[0]);
    }

    if (WRITE_STATISTICS_TO_FILE)
    {
        writeStatisticsToFile(stats, argv[0]);
    }

    freeAllocatedMemory(h_ca, headsWrite);
}