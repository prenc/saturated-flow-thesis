#include "hip/hip_runtime.h"
#include "../../common/memory_management.cuh"
#include "../../common/statistics.h"
#include <thrust/device_vector.h>


__global__ void simulation_step_kernel(CA ca, double *headsWrite, const int *activeCellsIdx, int
acNumber)
{
    unsigned ac_idx_x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned ac_idx_y = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned ac_idx_g = ac_idx_y * blockDim.x * gridDim.x + ac_idx_x;

    if (ac_idx_g < acNumber)
    {
        double Q{}, diff_head, tmp_t, ht1, ht2;
        unsigned idx_g = activeCellsIdx[ac_idx_g];
        unsigned idx_x = idx_g % COLS;
        unsigned idx_y = idx_g / COLS;
#ifdef LOOP
        for (int i = 0; i < KERNEL_LOOP_SIZE; i++)
        {
            if (i == KERNEL_LOOP_SIZE - 1)
            {
                if (Q) { Q = 0; }
            }
#endif
        if (idx_x >= 1)
        {
            diff_head = ca.heads[idx_g - 1] - ca.heads[idx_g];
            tmp_t = ca.K[idx_g] * THICKNESS;
            Q += diff_head * tmp_t;
        }
        if (idx_y >= 1)
        {
            diff_head = ca.heads[(idx_y - 1) * COLS + idx_x] - ca.heads[idx_g];
            tmp_t = ca.K[idx_g] * THICKNESS;
            Q += diff_head * tmp_t;
        }
        if (idx_x + 1 < COLS)
        {
            diff_head = ca.heads[idx_g + 1] - ca.heads[idx_g];
            tmp_t = ca.K[idx_g] * THICKNESS;
            Q += diff_head * tmp_t;
        }
        if (idx_y + 1 < ROWS)
        {
            diff_head = ca.heads[(idx_y + 1) * COLS + idx_x] - ca.heads[idx_g];
            tmp_t = ca.K[idx_g] * THICKNESS;
            Q += diff_head * tmp_t;
        }
#ifdef LOOP
        }
#endif
        Q -= ca.sources[idx_g];
        ht1 = Q * DELTA_T;
        ht2 = AREA * ca.Sy[idx_g];

        headsWrite[idx_g] = ca.heads[idx_g] + ht1 / ht2;
        if (headsWrite[idx_g] < 0)
        { headsWrite[idx_g] = 0; }
    }
}

__global__ void findActiveCells(struct CA d_ca, int *dv)
{
    unsigned idx_x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned idx_y = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned idx_g = idx_y * COLS + idx_x;

    if (idx_x < ROWS && idx_y < COLS)
    {
        if (d_ca.heads[idx_g] < INITIAL_HEAD || d_ca.sources[idx_g] != 0)
        {
            dv[idx_g] = idx_g;
            return;
        }
        if (idx_x > 0)
        {
            if (d_ca.heads[idx_g - 1] < INITIAL_HEAD)
            {
                dv[idx_g] = idx_g;
                return;
            }
        }
        if (idx_y > 0)
        {
            if (d_ca.heads[idx_g - COLS] < INITIAL_HEAD)
            {
                dv[idx_g] = idx_g;
                return;
            }
        }
        if (idx_x < COLS - 1)
        {
            if (d_ca.heads[idx_g + 1] < INITIAL_HEAD)
            {
                dv[idx_g] = idx_g;
                return;
            }
        }
        if (idx_y < ROWS - 1)
        {
            if (d_ca.heads[idx_g + COLS] < INITIAL_HEAD)
            {
                dv[idx_g] = idx_g;
                return;
            }
        }
        dv[idx_g] = 0;
    }
}

template<typename T>
struct is_non_zero
{
    __host__ __device__
    auto operator()(T x) const -> bool
    {
        return x != 0;
    }
};

int main(int argc, char *argv[])
{
    CA *d_ca = new CA();
    CA *h_ca = new CA();
    double *headsWrite;

    thrust::device_vector<int> dv(COLS * ROWS);
    thrust::device_vector<int> dv_p(COLS * ROWS);

    h_ca->heads = new double[ROWS * COLS]();
    h_ca->Sy = new double[ROWS * COLS]();
    h_ca->K = new double[ROWS * COLS]();
    h_ca->sources = new double[ROWS * COLS]();

    initializeCA(h_ca);

    allocateMemory(d_ca, headsWrite);
    copyDataFromCpuToGpu(h_ca, d_ca);
    ERROR_CHECK(hipMemcpy(headsWrite,
                           h_ca->heads, sizeof(double) * ROWS * COLS, hipMemcpyHostToDevice));

    dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
    const int blockCount = ceil((double) (ROWS * COLS) / (BLOCK_SIZE * BLOCK_SIZE));
    int gridSize = ceil(sqrt(blockCount));
    dim3 gridDims(gridSize, gridSize);

    std::vector<StatPoint> stats;
    Timer stepTimer, activeCellsEvalTimer, transitionTimer;
    stepTimer.start();

    bool isWholeGridActive = false;
    dim3 *simulationGridDims;
    int devActiveCellsCount;
    for (int i{}; i < SIMULATION_ITERATIONS; ++i)
    {
        if (!isWholeGridActive)
        {
            activeCellsEvalTimer.start();
            findActiveCells <<< gridDims, blockSize >>>(*d_ca, thrust::raw_pointer_cast(&dv[0]));
            ERROR_CHECK(hipDeviceSynchronize());
            activeCellsEvalTimer.stop();

            thrust::copy_if(thrust::device, dv.begin(), dv.end(), dv_p.begin(), is_non_zero<int>());

            devActiveCellsCount = thrust::count_if(dv_p.begin(), dv_p.end(), is_non_zero<int>());

            isWholeGridActive = devActiveCellsCount >= ROWS * COLS;

            int activeBlockCount = ceil((double) devActiveCellsCount / (BLOCK_SIZE * BLOCK_SIZE));
            int activeGridSize = ceil(sqrt(activeBlockCount));
            dim3 activeGridDim(activeGridSize, activeGridSize);

            simulationGridDims = &activeGridDim;
        }
        else
        {
            simulationGridDims = &gridDims;
        }

        transitionTimer.start();
        simulation_step_kernel <<< *simulationGridDims, blockSize >>>(
                *d_ca, headsWrite, thrust::raw_pointer_cast(&dv_p[0]), dv_p.size());
        hipDeviceSynchronize();
        transitionTimer.stop();

        double *tmpHeads = d_ca->heads;
        d_ca->heads = headsWrite;
        headsWrite = tmpHeads;

        if (i % STATISTICS_WRITE_FREQ == 0)
        {
            stepTimer.stop();
            auto stat = new StatPoint(
                    devActiveCellsCount / (double) (ROWS * COLS),
                    stepTimer.elapsedNanoseconds(),
                    transitionTimer.elapsedNanoseconds(),
                    activeCellsEvalTimer.elapsedNanoseconds());
            stats.push_back(*stat);
            stepTimer.start();
        }
    }

    if (WRITE_OUTPUT_TO_FILE)
    {
        copyDataFromGpuToCpu(h_ca, d_ca);
        saveHeadsInFile(h_ca->heads, argv[0]);
    }

    if (WRITE_STATISTICS_TO_FILE)
    {
        writeStatisticsToFile(stats, argv[0]);
    }

    freeAllocatedMemory(d_ca, headsWrite);
}