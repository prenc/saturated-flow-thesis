#include "hip/hip_runtime.h"
#include "../../common/memory_management.cuh"
#include "../../common/statistics.h"

__managed__ int dev_active_cells_count = 0;

__device__ unsigned active_cells_idx[ROWS * COLS];

__device__ void my_push_back(unsigned cellIdx)
{
    int insert_ptr = atomicAdd(&dev_active_cells_count, 1);
    active_cells_idx[insert_ptr] = cellIdx;
}

__global__ void simulation_step_kernel(struct CA *d_ca, double *d_write_head)
{
    int activeBlockCount = ceil((double) dev_active_cells_count / (BLOCK_SIZE * BLOCK_SIZE));
    int activeGridSize = ceil(sqrtf(activeBlockCount));

    unsigned ac_idx_x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned ac_idx_y = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned ac_idx_g = ac_idx_y * blockDim.x * activeGridSize + ac_idx_x;

    double Q, diff_head, tmp_t, ht1, ht2;
    if (ac_idx_g < dev_active_cells_count)
    {
        unsigned idx_g = active_cells_idx[ac_idx_g];
        unsigned idx_x = idx_g % COLS;
        unsigned idx_y = idx_g / COLS;

        if (idx_y != 0 && idx_y != ROWS - 1)
        {
            Q = 0;
            if (idx_x >= 1)
            {
                diff_head = d_ca->heads[idx_g - 1] - d_ca->heads[idx_g];
                tmp_t = d_ca->K[idx_g] * THICKNESS;
                Q += diff_head * tmp_t;
            }
            if (idx_y >= 1)
            {
                diff_head = d_ca->heads[(idx_y - 1) * COLS + idx_x] - d_ca->heads[idx_g];
                tmp_t = d_ca->K[idx_g] * THICKNESS;
                Q += diff_head * tmp_t;
            }
            if (idx_x + 1 < COLS)
            {
                diff_head = d_ca->heads[idx_g + 1] - d_ca->heads[idx_g];
                tmp_t = d_ca->K[idx_g] * THICKNESS;
                Q += diff_head * tmp_t;
            }
            if (idx_y + 1 < ROWS)
            {
                diff_head = d_ca->heads[(idx_y + 1) * COLS + idx_x] - d_ca->heads[idx_g];
                tmp_t = d_ca->K[idx_g] * THICKNESS;
                Q += diff_head * tmp_t;
            }

            Q -= d_ca->sources[idx_g];
            ht1 = Q * DELTA_T;
            ht2 = AREA * d_ca->Sy[idx_g];

            d_write_head[idx_g] = d_ca->heads[idx_g] + ht1 / ht2;
            if (d_write_head[idx_g] < 0)
            {
                d_write_head[idx_g] = 0;
            }
        }
    }
}

__global__ void find_active_cells_kernel(struct CA *d_ca)
{
    unsigned idx_x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned idx_y = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned idx_g = idx_y * COLS + idx_x;

    if (idx_x < ROWS && idx_y < COLS)
    {
        if (d_ca->heads[idx_g] < headFixed || d_ca->sources[idx_g] != 0)
        {
            my_push_back(idx_g);
            return;
        }
        if (idx_x >= 1)
        {
            if (d_ca->heads[idx_g - 1] < headFixed)
            {
                my_push_back(idx_g);
                return;
            }
        }
        if (idx_y >= 1)
        {
            if (d_ca->heads[idx_g - COLS] < headFixed)
            {
                my_push_back(idx_g);
                return;
            }
        }
        if (idx_x + 1 < COLS)
        {
            if (d_ca->heads[idx_g + 1] < headFixed)
            {
                my_push_back(idx_g);
                return;
            }
        }
        if (idx_y + 1 < ROWS)
        {
            if (d_ca->heads[idx_g + COLS] < headFixed)
            {
                my_push_back(idx_g);
                return;
            }
        }
    }
}

void perform_simulation_on_GPU()
{
    dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);

    const int blockCount = ceil((ROWS * COLS) / (BLOCK_SIZE * BLOCK_SIZE));
    int gridSize = ceil(sqrt(blockCount));
    dim3 gridDim(gridSize, gridSize);

    int activeBlockCount, activeGridSize;

    Timer stepTimer;
    bool isWholeGridActive = false;
    startTimer(&stepTimer);

    for (int i = 0; i < SIMULATION_ITERATIONS; i++)
    {
        dim3 *simulationGridDim;
        if (!isWholeGridActive)
        {

            dev_active_cells_count = 0;
            find_active_cells_kernel <<< gridDim, blockSize>>>(d_read_ca);
            hipDeviceSynchronize();

            isWholeGridActive = dev_active_cells_count == (ROWS * COLS);

            activeBlockCount = ceil((double) dev_active_cells_count / (BLOCK_SIZE * BLOCK_SIZE));
            activeGridSize = ceil(sqrt(activeBlockCount));
            dim3 activeGridDim(activeGridSize, activeGridSize);

            simulationGridDim = &activeGridDim;
        } else
        {
            simulationGridDim = &gridDim;
        }

        simulation_step_kernel <<< *simulationGridDim, blockSize >>>(d_read_ca, d_write_head);

        hipDeviceSynchronize();

        double *tmp1 = d_write_head;
        ERROR_CHECK(
                hipMemcpy(&d_write_head, &(d_read_ca->heads), sizeof(d_read_ca->heads), hipMemcpyDeviceToHost));
        ERROR_CHECK(hipMemcpy(&(d_read_ca->heads), &tmp1, sizeof(tmp1), hipMemcpyHostToDevice));

        if (i % STATISTICS_WRITE_FREQ == 0)
        {
            endTimer(&stepTimer);
            stats[i].coverage = double(dev_active_cells_count) / (ROWS * COLS);
            stats[i].stepTime = getElapsedTime(stepTimer);
            startTimer(&stepTimer);
        }
    }
}

int main(int argc, char *argv[])
{
    initializeCA();
    copyDataFromCpuToGpu();

    perform_simulation_on_GPU();

    if (WRITE_OUTPUT_TO_FILE)
    {
        copyDataFromGpuToCpu();
        write_heads_to_file(h_ca.heads, argv[0]);
    }

    write_statistics_to_file(argv[0]);

    return 0;
}

#pragma clang diagnostic pop