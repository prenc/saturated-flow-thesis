#include "hip/hip_runtime.h"
#include "../../params.h"
#include "../common/memory_management.cuh"
#include "../common/statistics.h"

__global__ void simulation_step_kernel(struct CA *d_ca, double *d_write_head) {
    __shared__ double s_heads[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ double s_K[BLOCK_SIZE][BLOCK_SIZE];
    unsigned idx_x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned idx_y = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned idx_g = idx_y * COLS + idx_x;

    double Q, diff_head, tmp_t, ht1, ht2;

    if (idx_x < COLS && idx_y < ROWS) {
        s_heads[threadIdx.y][threadIdx.x] = d_ca->head[idx_g];
        s_K[threadIdx.y][threadIdx.x] = d_ca->K[idx_g];
        __syncthreads();
        if (idx_y != 0 && idx_y != ROWS - 1) {
            for (int i = 0; i < KERNEL_LOOP_SIZE; i++) {
	            if (i == KERNEL_LOOP_SIZE - 1){
		            if (Q) {
			            Q = 0;
		            }
	            }
                if (idx_x >= 1) { // left neighbor
                    if (threadIdx.x >= 1)
                        diff_head = s_heads[threadIdx.y][threadIdx.x - 1] - s_heads[threadIdx.y][threadIdx.x];
                    else
                        diff_head = d_ca->head[idx_g - 1] - s_heads[threadIdx.y][threadIdx.x];
                    tmp_t = s_K[threadIdx.y][threadIdx.x] * THICKNESS;
                    Q += diff_head * tmp_t;
                }
                if (idx_y >= 1) { // upper neighbor
                    if (threadIdx.y >= 1)
                        diff_head = s_heads[threadIdx.y - 1][threadIdx.x] - s_heads[threadIdx.y][threadIdx.x];
                    else
                        diff_head = d_ca->head[(idx_y - 1) * COLS + idx_x] - s_heads[threadIdx.y][threadIdx.x];
                    tmp_t = s_K[threadIdx.y][threadIdx.x] * THICKNESS;
                    Q += diff_head * tmp_t;
                }
                if (idx_x + 1 < COLS) { // right neighbor
                    if (threadIdx.x < BLOCK_SIZE - 1)
                        diff_head = s_heads[threadIdx.y][threadIdx.x + 1] - s_heads[threadIdx.y][threadIdx.x];
                    else
                        diff_head = d_ca->head[idx_g + 1] - s_heads[threadIdx.y][threadIdx.x];
                    tmp_t = s_K[threadIdx.y][threadIdx.x] * THICKNESS;
                    Q += diff_head * tmp_t;
                }
                if (idx_y + 1 < ROWS) { // bottom neighbor
                    if (threadIdx.y < BLOCK_SIZE - 1)
                        diff_head = s_heads[threadIdx.y + 1][threadIdx.x] - s_heads[threadIdx.y][threadIdx.x];
                    else
                        diff_head = d_ca->head[(idx_y + 1) * COLS + idx_x] - s_heads[threadIdx.y][threadIdx.x];
                    tmp_t = s_K[threadIdx.y][threadIdx.x] * THICKNESS;
                    Q += diff_head * tmp_t;
                }
            }

            Q -= d_ca->Source[idx_g];
            ht1 = Q * DELTA_T;
            ht2 = AREA * d_ca->Sy[idx_g];

            d_write_head[idx_g] = s_heads[threadIdx.y][threadIdx.x] + ht1 / ht2;
	        if (d_write_head[idx_g] < 0) {
		        d_write_head[idx_g] = 0;
	        }
        }
    }
}

void perform_simulation_on_GPU() {
    dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
    const int blockCount = ceil((ROWS * COLS) / (BLOCK_SIZE * BLOCK_SIZE));
    double gridSize = ceil(sqrt(blockCount));
    dim3 gridDim(gridSize, gridSize);

    for (int i = 0; i < SIMULATION_ITERATIONS; i++) {
        simulation_step_kernel << < gridDim, blockSize >> > (d_read_ca, d_write_head);

        hipDeviceSynchronize();

        double *tmp1 = d_write_head;
        CUDASAFECALL(
                hipMemcpy(&d_write_head, &(d_read_ca->head), sizeof(d_read_ca->head), hipMemcpyDeviceToHost));
        CUDASAFECALL(hipMemcpy(&(d_read_ca->head), &tmp1, sizeof(tmp1), hipMemcpyHostToDevice));
    }
}

int main(int argc, char *argv[]) {
    init_host_ca();
    copy_data_from_CPU_to_GPU();

    perform_simulation_on_GPU();

	if(WRITE_OUTPUT_TO_FILE){
		copy_data_from_GPU_to_CPU();
		write_heads_to_file(h_ca.head, argv[0]);
	}

    return 0;
}
