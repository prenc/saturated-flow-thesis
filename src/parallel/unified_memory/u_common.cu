#include "hip/hip_runtime.h"
#include "../kernels/transition_kernels.cu"
#include "../common/statistics.h"

int main(int argc, char *argv[])
{
    auto h_ca = new CA();
    double *headsWrite;
    allocateManagedMemory(h_ca, headsWrite);

    initializeCA(h_ca);
    memcpy(headsWrite, h_ca->heads, sizeof(double) * ROWS * COLS);

    dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
    const int blockCount = ceil((double) (ROWS * COLS) / (BLOCK_SIZE * BLOCK_SIZE));
    int gridSize = ceil(sqrt(blockCount));
    dim3 gridDims(gridSize, gridSize);

    std::vector<StatPoint> stats;
    Timer stepTimer, transitionTimer;
    stepTimer.start();

    for (unsigned i{}; i < SIMULATION_ITERATIONS; ++i)
    {

#ifdef STANDARD
        kernels::standard_step <<< gridDims, blockSize >>>(*h_ca, headsWrite);
#endif
#ifdef HYBRID
        kernels::hybrid_step <<< gridDims, blockSize >>>(*h_ca, headsWrite);
#endif
#ifdef SHARED
        kernels::shared_step <<< gridDims, blockSize >>>(*h_ca, headsWrite);
#endif
        ERROR_CHECK(hipDeviceSynchronize());


        auto tmpHeads = h_ca->heads;
        h_ca->heads = headsWrite;
        headsWrite = tmpHeads;

        if (i % STATISTICS_WRITE_FREQ == STATISTICS_WRITE_FREQ - 1)
        {
            stepTimer.stop();
            auto stat = new StatPoint(
                    -1,
                    stepTimer.elapsedNanoseconds);
            stats.push_back(*stat);
            stepTimer.start();
        }
    }

    if (WRITE_OUTPUT_TO_FILE)
    {
        saveHeadsInFile(h_ca->heads, argv[0]);
    }

    if (WRITE_STATISTICS_TO_FILE)
    {
        writeStatisticsToFile(stats, argv[0]);
    }

    freeAllocatedMemory(h_ca, headsWrite);
}
