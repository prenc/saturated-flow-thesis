#include "hip/hip_runtime.h"
#include "../common/unified_memory_management.cuh"
#include "../common/statistics.h"

__global__ void simulation_step_kernel(struct CA d_ca, double *d_write_head) {
    unsigned x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned y = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned idx_g = y * blockDim.y * gridDim.x + x;

    double Q, diff_head, tmp_t, ht1, ht2;
    if (idx_g < ROWS * COLS) {
        unsigned idx_x = idx_g % COLS;
        unsigned idx_y = idx_g / COLS;

        if (idx_y != 0 && idx_y != ROWS - 1) {
            Q = 0;
            if (idx_x >= 1) {
                diff_head = d_ca.heads[idx_g - 1] - d_ca.heads[idx_g];
                tmp_t = d_ca.K[idx_g] * THICKNESS;
                Q += diff_head * tmp_t;
            }
            if (idx_y >= 1) {
                diff_head = d_ca.heads[(idx_y - 1) * COLS + idx_x] - d_ca.heads[idx_g];
                tmp_t = d_ca.K[idx_g] * THICKNESS;
                Q += diff_head * tmp_t;
            }
            if (idx_x + 1 < COLS) {
                diff_head = d_ca.heads[idx_g + 1] - d_ca.heads[idx_g];
                tmp_t = d_ca.K[idx_g] * THICKNESS;
                Q += diff_head * tmp_t;
            }
            if (idx_y + 1 < ROWS) {
                diff_head = d_ca.heads[(idx_y + 1) * COLS + idx_x] - d_ca.heads[idx_g];
                tmp_t = d_ca.K[idx_g] * THICKNESS;
                Q += diff_head * tmp_t;
            }

            Q -= d_ca.sources[idx_g];
            ht1 = Q * DELTA_T;
            ht2 = AREA * d_ca.Sy[idx_g];

            d_write_head[idx_g] = d_ca.heads[idx_g] + ht1 / ht2;
	        if (d_write_head[idx_g] < 0) {
		        d_write_head[idx_g] = 0;
	        }
        }
    }
}

void perform_simulation_on_GPU() {
    dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE);
    const int blockCount = ceil((ROWS * COLS) / (BLOCK_SIZE * BLOCK_SIZE));
    double gridSize = ceil(sqrt(blockCount));
    dim3 gridDim(gridSize, gridSize);

    Timer stepTimer;
    startTimer(&stepTimer);

    for (int i = 0; i < SIMULATION_ITERATIONS; i++) {

        simulation_step_kernel <<<gridDim, blockDim>>> (d_read, d_write.heads);
        hipDeviceSynchronize();

        double *tmp = d_write.heads;
        d_write.heads = d_read.heads;
        d_read.heads = tmp;

        if (i % STATISTICS_WRITE_FREQ == 0) {
            endTimer(&stepTimer);
            stats[i].stepTime = getElapsedTime(stepTimer);
            startTimer(&stepTimer);
        }
    }
}

int main(int argc, char *argv[]) {
    allocate_memory();
    init_read_ca();
    init_write_head();

    perform_simulation_on_GPU();

    if (WRITE_OUTPUT_TO_FILE){
        write_heads_to_file(d_write.heads, argv[0]);
    }

    if (WRITE_STATISTICS_TO_FILE) {
        write_statistics_to_file( argv[0]);
    }

    return 0;
}
