#include "hip/hip_runtime.h"
#include "unified_memory_common.h"
#include <sys/time.h>

double coverage_vector[ROWS*COLS];
double step_time_vector[ROWS*COLS];

__global__ void simulation_step_kernel(struct CA d_ca, double *d_write_head) {
    unsigned idx_x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned idx_y = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned idx_g = idx_y * COLS + idx_x;

    double Q, diff_head, tmp_t, ht1, ht2;
    if (idx_x < COLS && idx_y < ROWS) {
        if (idx_y != 0 && idx_y != ROWS - 1) {
            Q = 0;
            if (idx_x >= 1) {
                diff_head = d_ca.head[idx_g - 1] - d_ca.head[idx_g];
                tmp_t = d_ca.K[idx_g] * THICKNESS;
                Q += diff_head * tmp_t;
            }
            if (idx_y >= 1) {
                diff_head = d_ca.head[(idx_y - 1) * COLS + idx_x] - d_ca.head[idx_g];
                tmp_t = d_ca.K[idx_g] * THICKNESS;
                Q += diff_head * tmp_t;
            }
            if (idx_x + 1 < COLS) {
                diff_head = d_ca.head[idx_g + 1] - d_ca.head[idx_g];
                tmp_t = d_ca.K[idx_g] * THICKNESS;
                Q += diff_head * tmp_t;
            }
            if (idx_y + 1 < ROWS) {
                diff_head = d_ca.head[(idx_y + 1) * COLS + idx_x] - d_ca.head[idx_g];
                tmp_t = d_ca.K[idx_g] * THICKNESS;
                Q += diff_head * tmp_t;
            }

            Q -= d_ca.Source[idx_g];
            ht1 = Q * DELTA_T;
            ht2 = AREA * d_ca.Sy[idx_g];

            d_write_head[idx_g] = d_ca.head[idx_g] + ht1 / ht2;
        }
    }
}

void perform_simulation_on_GPU() {
    dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE);
    const int blockCount = ceil((ROWS * COLS) / (BLOCK_SIZE * BLOCK_SIZE));
    double gridSize = ceil(sqrt(blockCount));
    dim3 gridDim(gridSize, gridSize);

    struct timeval t1, t2;

    for (int i = 0; i < SIMULATION_ITERATIONS; i++) {
        gettimeofday(&t1, NULL);

        simulation_step_kernel <<< gridDim, blockDim >>> (d_read, d_write.head);

        hipDeviceSynchronize();

        double *tmp1 = d_write.head;
        d_write.head = d_read.head;
        d_read.head = tmp1;

        gettimeofday(&t2, NULL);

        step_time_vector[i] = t2.tv_usec - t1.tv_usec;
        coverage_vector[i] = 100;
    }
}

int main(int argc, char *argv[]) {
    allocate_memory();
    init_read_ca();
    init_write_head();

    perform_simulation_on_GPU();

    if (WRITE_OUTPUT_TO_FILE){
        write_heads_to_file(d_write.head, argv[0]);
    }

    if (WRITE_COVERAGE_TO_FILE) {
        write_coverage_to_file(coverage_vector, step_time_vector, argv[0]);
    }

    return 0;
}
