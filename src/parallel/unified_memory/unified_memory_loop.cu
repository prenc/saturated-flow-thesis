#include "hip/hip_runtime.h"
#include "unified_memory_common.h"

__global__ void simulation_step_kernel(struct CA d_ca, double *d_write_head, int gridSize) {
    unsigned x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned y = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned idx_g = y * blockDim.y * gridDim.x + x;

    double Q, diff_head, tmp_t, ht1, ht2;

    for (int i = 0; i < KERNEL_LOOP_SIZE; i++) {
        if (idx_g < ROWS * COLS) {
            unsigned idx_x = idx_g % COLS;
            unsigned idx_y = idx_g / COLS;

            Q = 0;
            if (idx_y != 0 && idx_y != ROWS - 1) {
                if (idx_x >= 1) {
                    diff_head = d_ca.head[idx_g - 1] - d_ca.head[idx_g];
                    tmp_t = d_ca.K[idx_g] * THICKNESS;
                    Q += diff_head * tmp_t;
                }
                if (idx_y >= 1) {
                    diff_head = d_ca.head[(idx_y - 1) * COLS + idx_x] - d_ca.head[idx_g];
                    tmp_t = d_ca.K[idx_g] * THICKNESS;
                    Q += diff_head * tmp_t;
                }
                if (idx_x + 1 < COLS) {
                    diff_head = d_ca.head[idx_g + 1] - d_ca.head[idx_g];
                    tmp_t = d_ca.K[idx_g] * THICKNESS;
                    Q += diff_head * tmp_t;
                }
                if (idx_y + 1 < ROWS) {
                    diff_head = d_ca.head[(idx_y + 1) * COLS + idx_x] - d_ca.head[idx_g];
                    tmp_t = d_ca.K[idx_g] * THICKNESS;
                    Q += diff_head * tmp_t;
                }
            }
            Q -= d_ca.Source[idx_g];
            ht1 = Q * DELTA_T;
            ht2 = AREA * d_ca.Sy[idx_g];

            d_write_head[idx_g] = d_ca.head[idx_g] + ht1 / ht2;
            if (d_write_head[idx_g] < 0) {
                d_write_head[idx_g] = 0;
            }
        }
    }
}

void perform_simulation_on_GPU() {
    dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
    const int blockCount = ceil((ROWS * COLS) / (BLOCK_SIZE * BLOCK_SIZE));
    double gridSize = ceil(sqrt(blockCount));
    dim3 blockCount2D(gridSize, gridSize);

    Timer stepTimer;
    startTimer(&stepTimer);

    for (int i = 0; i < SIMULATION_ITERATIONS; i++) {

        simulation_step_kernel << < blockCount2D, blockSize >> > (d_read, d_write.head, gridSize);
        hipDeviceSynchronize();

        double *tmp1 = d_write.head;
        d_write.head = d_read.head;
        d_read.head = tmp1;

        if (i % STATISTICS_WRITE_FREQ == 0) {
            endTimer(&stepTimer);
            stats[i].stepTime = getElapsedTime(stepTimer);
            startTimer(&stepTimer);
        }
    }
}

int main(int argc, char *argv[]) {
    allocate_memory();
    init_read_ca();
    init_write_head();

    perform_simulation_on_GPU();

    if(WRITE_OUTPUT_TO_FILE){
        write_heads_to_file(d_write.head, argv[0]);
    }

    if (WRITE_STATISTICS_TO_FILE) {
        write_statistics_to_file(stats, argv[0]);
    }

    return 0;
}

