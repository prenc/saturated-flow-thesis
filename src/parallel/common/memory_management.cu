#include "memory_management.cuh"

void allocateManagedMemory(CA *&ca, double *&heads_write)
{
    ERROR_CHECK(hipMallocManaged((void **) &heads_write, sizeof(double) * ROWS * COLS));
    ERROR_CHECK(hipMallocManaged((void **) &ca->heads, sizeof(double) * ROWS * COLS));
    ERROR_CHECK(hipMallocManaged((void **) &ca->K, sizeof(double) * ROWS * COLS));
    ERROR_CHECK(hipMallocManaged((void **) &ca->Sy, sizeof(double) * ROWS * COLS));
    ERROR_CHECK(hipMallocManaged((void **) &ca->sources, sizeof(double) * ROWS * COLS));
}

void allocateMemory(CA *&ca, double *&headsWrite)
{
    ERROR_CHECK(hipMalloc((void **) &headsWrite, sizeof(double) * ROWS * COLS));
    ERROR_CHECK(hipMalloc((void **) &ca->heads, sizeof(double) * ROWS * COLS));
    ERROR_CHECK(hipMalloc((void **) &ca->K, sizeof(double) * ROWS * COLS));
    ERROR_CHECK(hipMalloc((void **) &ca->Sy, sizeof(double) * ROWS * COLS));
    ERROR_CHECK(hipMalloc((void **) &ca->sources, sizeof(double) * ROWS * COLS));
}

void copyDataFromCpuToGpu(CA *&h_ca, CA *&d_ca)
{
    ERROR_CHECK(hipMemcpy(d_ca->heads, h_ca->heads, sizeof(double) * ROWS * COLS, hipMemcpyHostToDevice));
    ERROR_CHECK(hipMemcpy(d_ca->K, h_ca->K, sizeof(double) * ROWS * COLS, hipMemcpyHostToDevice));
    ERROR_CHECK(hipMemcpy(d_ca->Sy, h_ca->Sy, sizeof(double) * ROWS * COLS, hipMemcpyHostToDevice));
    ERROR_CHECK(hipMemcpy(d_ca->sources, h_ca->sources, sizeof(double) * ROWS * COLS, hipMemcpyHostToDevice));
}

CA *initializeCA()
{
    int wellsRows[] = {WELLS_Y};
    int wellsCols[] = {WELLS_X};
    double wellsQW[] = {WELLS_QW};

    auto ca = new CA();
    ca->heads = new double[ROWS * COLS]();
    ca->Sy = new double[ROWS * COLS]();
    ca->K = new double[ROWS * COLS]();
    ca->sources = new double[ROWS * COLS]();

    for (int i{}; i < ROWS; ++i)
        for (int j{}; j < COLS; ++j)
        {
            ca->heads[i * ROWS + j] = headFixed;
            if (j == COLS - 1)
            {
                ca->heads[i * ROWS + j] = headCalculated;
            }
            ca->Sy[i * ROWS + j] = Syinitial;
            ca->K[i * ROWS + j] = Kinitial;
            ca->sources[i * ROWS + j] = 0;
        }

    int x, y;
    double source;
    for (int i{}; i < NUMBER_OF_WELLS; ++i)
    {
        x = wellsRows[i];
        y = wellsCols[i];
        source = wellsQW[i];
        ca->sources[y * ROWS + x] = source;
    }
    return ca;
}

void copyDataFromGpuToCpu(CA *&h_ca, CA *&d_ca)
{
    ERROR_CHECK(hipMemcpy(h_ca->heads, d_ca->heads, sizeof(double) * ROWS * COLS, hipMemcpyDeviceToHost));
}

void free_allocated_memory(CA *&d_ca, double *&headsWrite)
{
    hipFree(headsWrite);
    hipFree(d_ca->heads);
    hipFree(d_ca->Sy);
    hipFree(d_ca->K);
    hipFree(d_ca->sources);
}
