#include "memory_management.cuh"

void allocateManagedMemory(CA *&ca, double *&heads_write)
{
    ERROR_CHECK(hipMallocManaged((void **) &heads_write, sizeof(double) * ROWS * COLS));
    ERROR_CHECK(hipMallocManaged((void **) &ca->heads, sizeof(double) * ROWS * COLS));
    ERROR_CHECK(hipMallocManaged((void **) &ca->K, sizeof(double) * ROWS * COLS));
    ERROR_CHECK(hipMallocManaged((void **) &ca->Sy, sizeof(double) * ROWS * COLS));
    ERROR_CHECK(hipMallocManaged((void **) &ca->sources, sizeof(double) * ROWS * COLS));
}

void allocateMemory(CA *&ca, double *&headsWrite)
{
    ERROR_CHECK(hipMalloc((void **) &headsWrite, sizeof(double) * ROWS * COLS));
    ERROR_CHECK(hipMalloc((void **) &ca->heads, sizeof(double) * ROWS * COLS));
    ERROR_CHECK(hipMalloc((void **) &ca->K, sizeof(double) * ROWS * COLS));
    ERROR_CHECK(hipMalloc((void **) &ca->Sy, sizeof(double) * ROWS * COLS));
    ERROR_CHECK(hipMalloc((void **) &ca->sources, sizeof(double) * ROWS * COLS));
}

void copyDataFromCpuToGpu(CA *&h_ca, CA *&d_ca)
{
    ERROR_CHECK(hipMemcpy(d_ca->heads, h_ca->heads, sizeof(double) * ROWS * COLS, hipMemcpyHostToDevice));
    ERROR_CHECK(hipMemcpy(d_ca->K, h_ca->K, sizeof(double) * ROWS * COLS, hipMemcpyHostToDevice));
    ERROR_CHECK(hipMemcpy(d_ca->Sy, h_ca->Sy, sizeof(double) * ROWS * COLS, hipMemcpyHostToDevice));
    ERROR_CHECK(hipMemcpy(d_ca->sources, h_ca->sources, sizeof(double) * ROWS * COLS, hipMemcpyHostToDevice));
}

void initializeCA(CA *&ca)
{
    int wellsRows[] = {WELLS_Y};
    int wellsCols[] = {WELLS_X};
    double wellsQW[] = {WELLS_QW};

    for (int i{}; i < ROWS; ++i)
    {
        for (int j{}; j < COLS; ++j)
        {
            ca->heads[i * ROWS + j] = INITIAL_HEAD;
            ca->Sy[i * ROWS + j] = INITIAL_SY;
            ca->K[i * ROWS + j] = INITIAL_K;
            ca->sources[i * ROWS + j] = 0;
        }
    }

    for (int i{}; i < NUMBER_OF_WELLS; ++i)
    {
        int x = wellsRows[i];
        int y = wellsCols[i];
        ca->sources[y * ROWS + x] = wellsQW[i];
    }
}

void copyDataFromGpuToCpu(CA *&h_ca, CA *&d_ca)
{
    ERROR_CHECK(hipMemcpy(h_ca->heads, d_ca->heads, sizeof(double) * ROWS * COLS, hipMemcpyDeviceToHost));
}

void freeAllocatedMemory(CA *&d_ca, double *&headsWrite)
{
    hipFree(headsWrite);
    hipFree(d_ca->heads);
    hipFree(d_ca->Sy);
    hipFree(d_ca->K);
    hipFree(d_ca->sources);
}
